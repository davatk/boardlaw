#include "hip/hip_runtime.h"
#include "../../cpp/kernels.cu"
#include "common.h"

namespace mctscuda {

const uint BLOCK = 8;

struct Policy {
    int A;
    float* pi;
    float* q;
    float lambda_n;
    float alpha;

    __device__ Policy(int A) :
        A(A)
    {
        extern __shared__ float shared[];
        pi = (float*)&shared[(2*threadIdx.x+0)*A];
        q  = (float*)&shared[(2*threadIdx.x+1)*A];
    }

    __device__ float prob(int a) {
        return lambda_n*pi[a]/(alpha - q[a]);
    }
    
    __host__ static uint memory(uint A) {
        uint mem = BLOCK*2*A*sizeof(float);
        TORCH_CHECK(mem < 64*1024, "Too much shared memory per block")
        return mem;
    }

};

__device__ float newton_search(Policy p) {
    // Find the initial alpha
    float alpha = 0.f;
    for (int a = 0; a<p.A; a++) {
        float gap = fmaxf(p.lambda_n*p.pi[a], 1.e-4f);
        alpha = fmaxf(alpha, p.q[a] + gap);
    }

    float error = HIP_INF_F;
    float new_error = HIP_INF_F;
    // Typical problems converge in 10 steps. Hypothetically 100 might be 
    // hit sometimes, but it's worth risking it for how utterly awful it'd 
    // be debugging an infinite loop in the kernel.
    for (int t=0; t<100; t++) {
        float S = 0.f; 
        float g = 0.f;
        for (int a=0; a<p.A; a++) {
            float top = p.lambda_n*p.pi[a];
            float bot = alpha - p.q[a];
            S += top/bot;
            g += -top/powf(bot, 2);
        }
        new_error = S - 1.f;
        // printf("%d: alpha: %.2f, S: %.2f, e: %.2f, g: %.2f\n", b, alpha, S, new_error, g);
        if ((new_error < 1e-3f) || (error == new_error)) {
            break;
        } else {
            alpha -= new_error/g;
            error = new_error;
        }
    }

    return alpha;
}

__device__ Policy policy(MCTSPTA m, H3D::PTA q, int t) {

    const uint A = m.logits.size(2);
    const int b = blockIdx.x*blockDim.x + threadIdx.x;

    Policy p(A);

    int N = 0;
    auto seat = m.seats[b][t];

    for (int a=0; a<p.A; a++) {
        auto child = m.children[b][t][a];

        if (child > -1) {
            p.q[a] = q[b][child][seat];
            p.pi[a] = expf(m.logits[b][t][a]);
            N += m.n[b][child];
        } else {
            p.q[a] = 0.f;
            p.pi[a] = expf(m.logits[b][t][a]);
            N += 1;
        }
    }
    __syncthreads(); // memory barrier

    p.lambda_n = m.c_puct[b]*sqrt(float(N))/float(N +A);
    p.alpha = newton_search(p);

    return p;
}

__host__ TT transition_q(MCTS m) {
    auto q = m.w.t/(m.n.t.unsqueeze(-1) + 1.e-4f);
    q = (q - q.min())/(q.max() - q.min() + 1.e-4f);
    return q.to(at::kHalf);
}

__global__ void root_kernel(MCTSPTA m, H3D::PTA q, H2D::PTA probs) {
    const uint B = m.logits.size(0);
    const uint A = m.logits.size(2);
    const int b = blockIdx.x*blockDim.x + threadIdx.x;
    if (b >= B) return;

    auto p = policy(m, q, 0);

    for (int a=0; a<A; a++) {
        probs[b][a] = p.prob(a);
    }
}

__host__ TT root(MCTS m) {
    c10::cuda::CUDAGuard g(m.logits.t.device());

    const uint B = m.logits.size(0);
    const uint A = m.logits.size(2);

    auto q = transition_q(m);

    auto probs = at::empty_like(m.logits.t.select(1, 0));

    const uint n_blocks = (B + BLOCK - 1)/BLOCK;
    root_kernel<<<{n_blocks}, {BLOCK}, Policy::memory(A), stream()>>>(
        m.pta(), H3D(q).pta(), H2D(probs).pta());
    C10_CUDA_CHECK(hipGetLastError());

    return probs;
}

__global__ void descend_kernel(
    MCTSPTA m, H3D::PTA q, H2D::PTA rands, DescentPTA descent) {

    const uint B = m.logits.size(0);
    const uint A = m.logits.size(2);
    const int b = blockIdx.x*blockDim.x + threadIdx.x;

    if (b >= B) return;

    int t = 0;
    int parent = 0;
    int action = -1;
    int valid = -1;
    while (true) {
        if (t == -1) break;
        if (m.terminal[b][t]) break;

        auto p = policy(m, q, t);

        float rand = rands[b][t];
        float total = 0.f;
        // This is a bit of a mess. Intent is to handle the edge 
        // case of rand being 1, and the probabilities not summing
        // to that. Then we need to fall back to a 'valid' value, 
        // ie one that has a positive probability.
        action = -1; 
        valid = -1;
        for (int a=0; a<A; a++) {
            float prob = p.prob(a);
            total += prob;
            if ((prob > 0) && (total >= rand)) {
                action = a;
                break;
            } else if (prob > 0) {
                valid = a;
            }
        }
        action = (action >= 0)? action : valid;
        parent = t;
        t = m.children[b][t][action];
    }

    descent.parents[b] = parent;
    descent.actions[b] = action;
}

__host__ Descent descend(MCTS m) {
    c10::cuda::CUDAGuard g(m.logits.t.device());

    const uint B = m.logits.size(0);
    const uint A = m.logits.size(2);

    auto q = transition_q(m);
    auto rands = at::rand_like(m.logits.t.select(2, 0));

    Descent descent{
        m.seats.t.new_empty({B}),
        m.seats.t.new_empty({B})};

    const uint n_blocks = (B + BLOCK - 1)/BLOCK;
    descend_kernel<<<{n_blocks}, {BLOCK}, Policy::memory(A), stream()>>>(
        m.pta(), H3D(q).pta(), H2D(rands).pta(), descent.pta());
    C10_CUDA_CHECK(hipGetLastError());

    return descent;
}

__global__ void backup_kernel(BackupPTA bk, S1D::PTA leaves) {
    const uint B = bk.v.size(0);
    const uint S = bk.v.size(2);
    const int b = blockIdx.x*blockDim.x + threadIdx.x;

    if (b >= B) return;

    extern __shared__ float shared[];
    float* v = (float*)&shared[threadIdx.x*S];

    int current = leaves[b];
    for (int s=0; s<S; s++) {
        v[s] = bk.v[b][current][s];
    }
    while (true) {
        if (current == -1) break;

        //TODO: Should invert this loop for memory locality buuuuut
        // it's not gonna be the bottleneck anyway. 
        for (int s=0; s<S; s++) {
            if (bk.terminal[b][current]) {
                v[s] = 0.f;
            }
            v[s] += bk.rewards[b][current][s];

            bk.n[b][current] += 1;
            bk.w[b][current][s] += v[s];
        }

        current = bk.parents[b][current]; 
    }
}

__host__ void backup(Backup bk, TT leaves) {
    c10::cuda::CUDAGuard c(leaves.device());

    const uint B = bk.v.size(0);
    const uint S = bk.v.size(2);

    const uint n_blocks = (B + BLOCK - 1)/BLOCK;
    backup_kernel<<<{n_blocks}, {BLOCK}, BLOCK*S*sizeof(float), stream()>>>(
        bk.pta(), S1D(leaves).pta());
    C10_CUDA_CHECK(hipGetLastError());
}

}